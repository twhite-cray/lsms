#include "hip/hip_runtime.h"
// -*- mode: c++; -*-

#include <stdlib.h>
#include "Real.hpp"
#include "Complex.hpp"
#include "Matrix.hpp"

#include "DeviceMatrix.hpp"
#include "DeviceArray3d.hpp"
#include "DeviceVector.hpp"
#include "Main/SystemParameters.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <iostream>

#ifdef _OPENMP
#include <omp.h>
#else
#ifndef LSMS_DUMMY_OPENMP
#define LSMS_DUMMY_OPENMP
inline int omp_get_max_threads() {return 1;}
inline int omp_get_num_threads() {return 1;}
inline int omp_get_thread_num() {return 0;}
#endif
#endif

#include "DeviceStorage.hpp"

// #include "cuda_error.h"
#include "deviceCheckError.hpp"

using namespace std;

//TODO move inside DeviceStorage?
//allocate a thread specific matrix on the host and pin its memory
extern "C"
Complex* get_host_m_(const int &max_nrmat_ns) {
  static Complex * m_v=0;
  static int cur_size=0;
  static hipError_t pinned;

  if(cur_size<max_nrmat_ns) {

    //release previously allocated memory
    if(m_v!=0) {
      if(pinned) hipHostFree(m_v);
      else free(m_v);
    }

    //allocate new memory
    pinned = hipHostMalloc((void**)&m_v,max_nrmat_ns*max_nrmat_ns*sizeof(Complex)*omp_get_max_threads());

    if ( pinned != hipSuccess )
    {
      fprintf(stderr, "Matrix not pinned\n");
      m_v = (Complex*)malloc(max_nrmat_ns*max_nrmat_ns*sizeof(Complex)*omp_get_max_threads());
    }
    cur_size=max_nrmat_ns;
  }
  return m_v; 
}

/*
static const int MAX_THREADS=16;
class DeviceStorage {
private:
  static int nThreads;
  static Complex *dev_m[MAX_THREADS], *dev_bgij[MAX_THREADS], *dev_tmat_n[MAX_THREADS];
  static Complex *dev_tau[MAX_THREADS], *dev_tau00[MAX_THREADS];
  static int *dev_ipvt[MAX_THREADS];
  static hipblasHandle_t hipblas.h[MAX_THREADS];
  static hipsolverHandle_t cusolverDnHandle[MAX_THREADS];
  static hipEvent_t event[MAX_THREADS];
  static hipStream_t stream[MAX_THREADS][2];
  static size_t dev_workBytes[MAX_THREADS];
  static void *dev_work[MAX_THREADS];
  static DeviceMatrix<Complex> dev_tmat_store;
  static bool initialized;
public:
*/

  int DeviceStorage::allocate(int kkrsz_max,int nspin, int numLIZ, int _nThreads)
  {
    if(!initialized)
    {
      //printf("*************************************MEMORY IS BEING ALLOCATED\n");
      if(_nThreads>MAX_THREADS)
      {
        printf("nThreads (%d) in DeviceStorage::allocate exceeds MAX_THREADS (%d)\n",_nThreads,MAX_THREADS);
        printf("  change MAX_THREADS in src/Accelerator/DeviceStorage.cu and recompile!\n");
        exit(1);
      }
      nThreads=_nThreads;
      int N=kkrsz_max*nspin*numLIZ;
      // printf("DeviceStorage::alocate N=%d\n",N);
      for(int i=0;i<nThreads;i++)
      {
        hipError_t err;
        err = hipMalloc((void**)&dev_m[i],N*N*sizeof(Complex));
        if(err!=hipSuccess)
        {
          printf("failed to allocate dev_m[%d], size=%d, err=%d\n",
                i,N*N*sizeof(Complex),err);
          exit(1);
        }
        hipMalloc((void**)&dev_ipvt[i],N*sizeof(int));
        hipMalloc((void**)&dev_info[i],nThreads*sizeof(int));
	err = hipMalloc((void**)&dev_bgij[i],N*N*sizeof(Complex));
        if(err!=hipSuccess)
        {
          printf("failed to allocate dev_bgij[%d], size=%d, err=%d\n",
                i,N*N*sizeof(Complex),err);
          exit(1);
        }
#ifdef BUILDKKRMATRIX_GPU
        // hipMalloc((void**)&dev_bgij[i],4*kkrsz_max*kkrsz_max*numLIZ*numLIZ*sizeof(Complex));
        hipMalloc((void**)&dev_tmat_n[i],4*kkrsz_max*kkrsz_max*numLIZ*sizeof(Complex)); 
#endif
        hipMalloc((void**)&dev_tau[i], 4*N*kkrsz_max*sizeof(Complex));
        hipMalloc((void**)&dev_tau00[i], 4*kkrsz_max*kkrsz_max*sizeof(Complex));
        hipMalloc((void**)&dev_t[i], 4*N*kkrsz_max*sizeof(Complex));
        hipMalloc((void**)&dev_t0[i], 4*kkrsz_max*kkrsz_max*sizeof(Complex));
        hipStreamCreate(&stream[i][0]);
        hipStreamCreate(&stream[i][1]);
        hipEventCreateWithFlags(&event[i],hipEventDisableTiming);
        hipblasCreate(&cublas_h[i]);
        hipsolverDnCreate(&cusolverDnHandle[i]);

	int lWork;
	hipsolverDnZgetrf_bufferSize(cusolverDnHandle[i], N, N,
				    (hipDoubleComplex *)dev_m[i], N, &lWork);
	dev_workBytes[i] = 0;
#ifndef ARCH_IBM
	hipsolverDnZZgesv_bufferSize(cusolverDnHandle[i], N, 2*kkrsz_max,
				    (hipDoubleComplex *)dev_m[i], N, dev_ipvt[i], (hipDoubleComplex *)dev_tau[i], N, (hipDoubleComplex *)dev_tau[i], N,
				    dev_work[i], &dev_workBytes[i]);
#endif

	dev_workBytes[i] = std::max(dev_workBytes[i]*sizeof(hipDoubleComplex),
				    lWork*sizeof(hipDoubleComplex));
	hipMalloc((void**)&dev_work[i], dev_workBytes[i]);
        // printf("  dev_m[%d]=%zx\n",i,dev_m[i]);
      }
      deviceCheckError();
      initialized=true;
    }
    return 0;
  }
  
  void DeviceStorage::free()
  {
    if(initialized) {
   //     printf("*************************************MEMORY IS BEING FREED\n");
      // for(int i=0;i<omp_get_max_threads();i++)
      for(int i=0; i<nThreads; i++)
      {
        hipFree(dev_m[i]);
        hipFree(dev_ipvt[i]);
        hipFree(dev_info[i]);
#ifdef BUILDKKRMATRIX_GPU
        hipFree(dev_bgij[i]);
        hipFree(dev_tmat_n[i]);
#endif
	hipFree(dev_work[i]);
        hipFree(dev_t0[i]);
        hipStreamDestroy(stream[i][0]);
        hipStreamDestroy(stream[i][1]);
        hipEventDestroy(event[i]);
        hipblasDestroy(cublas_h[i]);
      }
      // dev_tmat_store.clear();
      hipFree(devTmatStore);
      deviceCheckError();
      initialized=false;
    }
  }

/*
  static Complex* getDevM() { return dev_m[omp_get_thread_num()]; } 
  static Complex* getDevBGij() { if(!initialized) {printf("DeviceStorage not initialized\n"); exit(1);}
                                 return dev_bgij[omp_get_thread_num()]; } 
  static Complex* getDevTmatN() { return dev_tmat_n[omp_get_thread_num()]; } 
  static Complex* getDevTau() { return dev_tau[omp_get_thread_num()]; }
  static Complex* getDevTau00() { return dev_tau00[omp_get_thread_num()]; }
  static int* getDevIpvt() { return dev_ipvt[omp_get_thread_num()]; } 
  static hipStream_t getStream(int i) { return stream[omp_get_thread_num()][i]; }
  static hipEvent_t getEvent() { return event[omp_get_thread_num()]; }
  static hipblasHandle_t getCublasHandle() { return hipblas.h[omp_get_thread_num()]; }
  static hipsolverHandle_t getCusolverDnHandle() { return cusolverDnHandle[omp_get_thread_num()]; }
  static size_t getDevWorkBytes() { return dev_workBytes[omp_get_thread_num()]; }
  static void *getDevWork() {  return dev_work[omp_get_thread_num()]; }
  static DeviceMatrix<Complex>* getDevTmatStore() { return &dev_tmat_store; }
};
*/

int DeviceStorage::copyTmatStoreToDevice(Matrix<Complex> &tmatStore,
    int blkSize)
{
  if((tmatStoreSize > 0) && (tmatStoreSize < tmatStore.size()))
  {
    hipFree(devTmatStore);
    tmatStoreSize = 0;
  }
  if(tmatStoreSize == 0)
  {
    hipMalloc(&devTmatStore, tmatStore.size()*sizeof(Complex));
    tmatStoreSize = tmatStore.size();
  }
  hipMemcpy(devTmatStore, &tmatStore(0,0),
    tmatStore.size()*sizeof(Complex), hipMemcpyHostToDevice);
  blkSizeTmatStore = blkSize;
  tmatStoreLDim = tmatStore.l_dim();

  return 0;
}

bool DeviceStorage::initialized = false;
Complex *DeviceStorage::dev_m[MAX_THREADS], *DeviceStorage::dev_bgij[MAX_THREADS], *DeviceStorage::dev_tmat_n[MAX_THREADS];
Complex *DeviceStorage::dev_tau[MAX_THREADS], *DeviceStorage::dev_tau00[MAX_THREADS];
Complex *DeviceStorage::dev_t0[MAX_THREADS];
Complex *DeviceStorage::dev_t[MAX_THREADS];
void *DeviceStorage::dev_work[MAX_THREADS];
size_t DeviceStorage::dev_workBytes[MAX_THREADS];
int *DeviceStorage::dev_ipvt[MAX_THREADS];
int *DeviceStorage::dev_info[MAX_THREADS];
hipblasHandle_t DeviceStorage::hipblas.h[MAX_THREADS];
hipsolverHandle_t DeviceStorage::cusolverDnHandle[MAX_THREADS];
hipEvent_t DeviceStorage::event[MAX_THREADS];
hipStream_t DeviceStorage::stream[MAX_THREADS][2];
// DeviceMatrix<Complex> DeviceStorage::dev_tmat_store;
Complex *DeviceStorage::devTmatStore;
size_t DeviceStorage::tmatStoreSize = 0;
int DeviceStorage::blkSizeTmatStore = 0;
int DeviceStorage::tmatStoreLDim = 0;
int DeviceStorage::nThreads=1;
bool initialized = false;

std::vector<DeviceAtom> deviceAtoms;

// Device Atom
int DeviceAtom::allocate(int _lmax, int _nspin, int _numLIZ)
{
  if(allocated) free();
  allocated = true;
  numLIZ = _numLIZ;
  hipMalloc((void**)&LIZPos,numLIZ*3*sizeof(Real));
  hipMalloc((void**)&LIZlmax,numLIZ*sizeof(int));
  hipMalloc((void**)&LIZStoreIdx,numLIZ*sizeof(int));
  
  return 0;
}

void DeviceAtom::free()
{
  if(allocated)
  {
    hipFree(LIZPos);
    hipFree(LIZlmax);
    hipFree(LIZStoreIdx);
  }
  allocated = false;
}

void DeviceAtom::copyFromAtom(AtomData &atom)
{
  if(!allocated)
  {
    allocate(atom.lmax, atom.nspin, atom.numLIZ);
  }
  hipMemcpy(LIZPos, &atom.LIZPos(0,0), atom.numLIZ*3*sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy(LIZlmax, &atom.LIZlmax[0], atom.numLIZ*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(LIZStoreIdx, &atom.LIZStoreIdx[0], atom.numLIZ*sizeof(int), hipMemcpyHostToDevice);
}

int *DeviceConstants::lofk;
int *DeviceConstants::mofk;
hipDoubleComplex *DeviceConstants::ilp1;
// DeviceMatrix<Complex> illp(ndlj, ndlj);
hipDoubleComplex* DeviceConstants::illp;
int DeviceConstants::ndlj_illp;
// DeviceArray3d<Real> cgnt(lmax+1,ndlj,ndlj);
Real* DeviceConstants::cgnt;
int DeviceConstants::ndlj_cgnt, DeviceConstants::lmaxp1_cgnt;

int DeviceConstants::allocate(AngularMomentumIndices &am, GauntCoeficients &c, IFactors &ifactors)
{
  ndlj_illp = ifactors.illp.l_dim();
  lmaxp1_cgnt = c.cgnt.l_dim1();
  ndlj_cgnt = c.cgnt.l_dim2();

  hipMalloc((void**)&lofk, am.lofk.size()*sizeof(int));
  hipMalloc((void**)&mofk, am.mofk.size()*sizeof(int));
  hipMalloc((void**)&ilp1, ifactors.ilp1.size()*sizeof(hipDoubleComplex));
  hipMalloc((void**)&illp, ifactors.illp.size()*sizeof(hipDoubleComplex));
  hipMalloc((void**)&cgnt, c.cgnt.size()*sizeof(double));

  hipMemcpy(lofk, &am.lofk[0], am.lofk.size()*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mofk, &am.mofk[0], am.mofk.size()*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(ilp1, &ifactors.ilp1[0], ifactors.ilp1.size()*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  hipMemcpy(illp, &ifactors.illp[0], ifactors.illp.size()*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  hipMemcpy(cgnt, &c.cgnt[0], c.cgnt.size()*sizeof(double), hipMemcpyHostToDevice);

  return 0;
}

void DeviceConstants::free()
{
  hipFree(lofk);
  hipFree(mofk);
  hipFree(ilp1);
  hipFree(illp);
  hipFree(cgnt);
}


/****************Fortran Interfaces*********************/
extern "C"
Complex* get_dev_m_() {
  return DeviceStorage::getDevM();
}

extern "C"
Complex* get_dev_bgij_() {
  return DeviceStorage::getDevBGij();
}

extern "C"
Complex* get_dev_tmat_n_() {
  return DeviceStorage::getDevTmatN();
}

extern "C"
int* get_dev_ipvt_() {
  return DeviceStorage::getDevIpvt();
}

extern "C"
hipStream_t get_stream_(const int &id) {
  return DeviceStorage::getStream(id);
}

extern "C"
hipblasHandle_t get_cublas_handle_() {
  return DeviceStorage::getCublasHandle();
}

//allocate a thread specific event
extern "C"
hipEvent_t get_cuda_event_() {
  return DeviceStorage::getEvent();
}
/********************************************************/

// DeviceMatrix<Complex>* get_dev_tmat_store() {
//   return DeviceStorage::getDevTmatStore();
// }

void *allocateDStore(void)
{
  return static_cast<void *>(new DeviceStorage);
}

void freeDStore(void * d_store)
{
  static_cast<DeviceStorage*>(d_store)->free();
  delete static_cast<DeviceStorage*>(d_store);
}

int initDStore(void * d_store,int kkrsz_max, int nspin, int numLIZ, int nthreads)
{
  return (*static_cast<DeviceStorage*>(d_store)).allocate(kkrsz_max,nspin,numLIZ,nthreads);
}

// void copyTmatStoreToDevice(LocalTypeInfo &local) {
//  DeviceMatrix<Complex> &d_tmat_store=*get_dev_tmat_store();
//  d_tmat_store.copy_async(local.tmatStore,0);
// }
